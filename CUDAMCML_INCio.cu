#include "hip/hip_runtime.h"
/*	This file is part of CUDAMCML_INC.

    CUDAMCML_INC is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCML_INC is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCML_INC.  If not, see <http://www.gnu.org/licenses/>.*/

#define NFLOATS 9
#define NINTS 5

#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <stdlib.h>
#include <float.h>
#include <string.h>

int interpret_arg(int argc, char* argv[], unsigned long long* seed, int* ignoreAdetection)
{

	int unknown_argument;
	for(int i=2;i<argc;i++)
	{
		unknown_argument=1;
		if(!strcmp(argv[i],"-A")) 
		{
			unknown_argument=0;
			*ignoreAdetection=1; //This option is not yet implemented. Therefore, this option has no effect.
			printf("Ignoring A-detection!\n");
		}
		if(!strncmp(argv[i],"-S",2) && sscanf(argv[i],"%*2c %llu",seed))
		{
		unknown_argument=0;
		printf("Seed=%llu\n",*seed);
		}
		if(unknown_argument)
		{
			printf("Unknown argument %s!\n",argv[i]);
			return 1;
		}
	}
	return 0;
}

int Write_Simulation_Results(MemStruct* HostMem, SimulationStruct* sim, clock_t simulation_time)
{
	FILE* pFile_inp;
	char mystring[STR_LEN];

	// Copy stuff from sim->det to make things more readable:
	double dx=(double)sim->det.dx;		// Detection grid resolution, x-direction [cm]
	double dy=(double)sim->det.dy;		// Detection grid resolution, y-direction [cm]
	
	int nx=sim->det.nx;			// Number of grid elements in x-direction
	int ny=sim->det.ny;			// Number of grid elements in y-direction
	

	int x,y;//,z;
	//unsigned int l;
	int i;

	//unsigned long long temp=0;
	double scale1 = (double)0xFFFFFFFF*(double)sim->number_of_photons; // Number of photons (used to normalize)
	double scale2;

	


		
	// Open the input and output files
	pFile_inp = fopen (sim->inp_filename , "r");
	if (pFile_inp == NULL){perror ("Error opening input file");return 0;}

	//pFile_outp = fopen (sim->outp_filename , "w");
	//if (pFile_outp == NULL){perror ("Error opening output file");return 0;}

	// Write other stuff here first!

	char salida_simdata[256];
	strcpy(salida_simdata, sim->outp_filename);
	strcat(salida_simdata,"_simdata.dat");

	char salida_trans[256];
	strcpy(salida_trans, sim->outp_filename);
	strcat(salida_trans,"_trans.dat");

	char salida_ref[256];
	strcpy(salida_ref, sim->outp_filename);
	strcat(salida_ref,"_ref.dat");

	
	char salida_banana[256];
	strcpy(salida_banana, sim->outp_filename);
	strcat(salida_banana,"_banana.dat");

	char salida_t[256];
	strcpy(salida_t, sim->outp_filename);
	strcat(salida_t,"_temp.dat");
	
	FILE* dFile_out;
	dFile_out = fopen (salida_simdata,"w");

	fprintf(dFile_out,"A1 	# Version number of the file format.\n\n");
	fprintf(dFile_out,"####\n");
	fprintf(dFile_out,"# Data categories include: \n");
	fprintf(dFile_out,"# InParm, RAT, \n");
	fprintf(dFile_out,"# A_l, A_z, Rd_r, Rd_a, Tt_r, Tt_a, \n");
	fprintf(dFile_out,"# A_rz, Rd_ra, Tt_ra \n");
	fprintf(dFile_out,"####\n\n");

	// Write simulation time
	fprintf(dFile_out,"# User time: %.2f sec\n\n",(double)simulation_time/CLOCKS_PER_SEC);


	fprintf(dFile_out,"InParam\t\t# Input parameters:\n");
	// Copy the input data from inp_filename
	fseek(pFile_inp, sim->begin, SEEK_SET);
	while(sim->end>ftell(pFile_inp))
	{
		
		fgets(mystring , STR_LEN , pFile_inp);
		fputs(mystring , dFile_out );
	}

	
	fclose(pFile_inp);
	fclose(dFile_out);
	
	FILE* trFile_out;
	trFile_out = fopen (salida_trans,"w");
	
	i=0;
	//fprintf(pFile_outp,"\n\n# T[x][y]. [1/(cm2)].\n# T[0][0], [0][1],..[0][ny-1]\n# T[1][0], [1][1],..[1][ny-1]\n# ...\n# T[nx-1][0], [nx-1][1],..[nx-1][ny-1]\nT\n");
	for(y=0;y<ny;y++)
	{
		for(x=0;x<nx;x++)
		{
			scale2=scale1*dx*dy; // Normalization Constant
			fprintf(trFile_out," %E ",(double)HostMem->Tt_xy[y*nx+x]/scale2);
		}
		fprintf(trFile_out," \n ");
	}

	fclose(trFile_out);

	FILE* reFile_out;
	reFile_out = fopen (salida_ref,"w");

	i=0;
	//fprintf(pFile_outp,"\n\n# R[x][y]. [1/(cm2)].\n# R[0][0], [0][1],..[0][ny-1]\n# R[1][0], [1][1],..[1][ny-1]\n# ...\n# R[nx-1][0], [nx-1][1],..[nx-1][ny-1]\nR\n");
	for(y=0;y<ny;y++)
	{
		for(x=0;x<nx;x++)
		{
			scale2=scale1*dx*dy; // Normalization Constant
			fprintf(reFile_out," %E ",(double)HostMem->Rd_xy[y*nx+x]/scale2);
		}
		fprintf(reFile_out," \n ");
	}

	fclose(reFile_out);

		
	
	FILE* bFile_out;
	bFile_out = fopen (salida_banana,"w");
	int max_z=(int)(sim->esp)*TAM_GRILLA;
	int max_x=(int)2*(sim->esp)*TAM_GRILLA;	
	for(int ix=0;ix<max_x;ix++){                                            /*Reconvierte i y j a x y z*/
        	for(int jz=0;jz<max_z;jz++){
            		fprintf(bFile_out, "%E\t", (double)HostMem->banana[ix*max_z+jz]/scale1);
        	}
        	fprintf(bFile_out,"\n");
    	}
	fclose (bFile_out);
	
	FILE* tFile_out;
	tFile_out = fopen (salida_t,"w");
	for(int itemp=0;itemp<NUM_CAN_TEMP;itemp++){                                            /*Reconvierte i y j a x y z*/
        		fprintf(tFile_out, " %E\t", (double)HostMem->histo_temp[itemp]/scale1);
        	fprintf(tFile_out,"\n");
    	}
	fclose (tFile_out);
	
	return 0;

}


int isnumeric(char a)
{
	if(a>=(char)48 && a<=(char)57) return 1;
	else return 0;
}

int readfloats(int n_floats, float* temp, FILE* pFile)
{
	int ii=0;
	char mystring [200];

	if(n_floats>NFLOATS) return 0; //cannot read more than NFLOATS floats

	while(ii<=0)
	{
		if(feof(pFile)) return 0; //if we reach EOF here something is wrong with the file!
		fgets(mystring , 200 , pFile);
		memset(temp,0,NFLOATS*sizeof(float));
		ii=sscanf(mystring,"%f %f %f %f %f %f %f %f %f",&temp[0],&temp[1],&temp[2],&temp[3],&temp[4],&temp[5],&temp[6],&temp[7],&temp[8]);
		if(ii>n_floats) return 0; //if we read more number than defined something is wrong with the file!
	}
	return 1; // Everyting appears to be ok!
}

int readints(int n_ints, int* temp, FILE* pFile) //replace with template?
{
	int ii=0;
	char mystring[STR_LEN];

	if(n_ints>NINTS) return 0; //cannot read more than NFLOATS floats

	while(ii<=0)
	{
		if(feof(pFile)) return 0; //if we reach EOF here something is wrong with the file!
		fgets(mystring , STR_LEN , pFile);
		memset(temp,0,NINTS*sizeof(int));
		ii=sscanf(mystring,"%d %d %d %d %d",&temp[0],&temp[1],&temp[2],&temp[3],&temp[4]);
		if(ii>n_ints) return 0; //if we read more number than defined something is wrong with the file!
	}
	return 1; // Everyting appears to be ok!
}

int ischar(char a)
{
	if((a>=(char)65 && a<=(char)90)||(a>=(char)97 && a<=(char)122)) return 1;
	else return 0;
}

int read_simulation_data(char* filename, SimulationStruct** simulations, int ignoreAdetection)
{
	int i=0;
	int ii=0;
	int iii=0;
	unsigned long number_of_photons;
	int n_inclusions = 0;
	unsigned int start_weight;
	int n_simulations = 0;
	int n_layers = 0;
	FILE * pFile;
	char mystring [STR_LEN];
	char str[STR_LEN];
	char AorB;
	float dtot=0;


	float ftemp[NFLOATS];//Find a more elegant way to do this...
	int itemp[NINTS];


	pFile = fopen(filename , "r");
	if (pFile == NULL){perror ("Error opening file");return 0;}
	
	// First read the first data line (file version) and ignore
	if(!readfloats(1, ftemp, pFile)){perror ("Error reading file version");return 0;}
	//printf("File version: %f\n",ftemp[0]);

	// Second, read the number of runs
	if(!readints(1, itemp, pFile)){perror ("Error reading number of runs");return 0;}
	n_simulations = itemp[0];
	printf("Number of runs: %d\n",n_simulations);
	
	// Allocate memory for the SimulationStruct array
	*simulations = (SimulationStruct*) malloc(sizeof(SimulationStruct)*n_simulations);
	if(*simulations == NULL){perror("Failed to malloc simulations.\n");return 0;}//{printf("Failed to malloc simulations.\n");return 0;}

	for(i=0;i<n_simulations;i++)
	{
		// Store the input filename
		strcpy((*simulations)[i].inp_filename,filename);
		// Echo the Filename
		//printf("Input filename: %s\n",filename);

		// Store ignoreAdetection data
		(*simulations)[i].ignoreAdetection=ignoreAdetection;

		// Read the output filename and determine ASCII or Binary output
		ii=0;
		while(ii<=0)
		{
			(*simulations)[i].begin=ftell(pFile);
			fgets (mystring , STR_LEN , pFile);
			ii=sscanf(mystring,"%s %c",str,&AorB);
			if(feof(pFile)|| ii>2){perror("Error reading output filename");return 0;}
			if(ii>0)ii=ischar(str[0]);
		}
		// Echo the Filename and AorB
		//printf("Output filename: %s, AorB=%c\n",str,AorB);
		strcpy((*simulations)[i].outp_filename,str);
		(*simulations)[i].AorB=AorB;

		//printf("begin=%d\n",(*simulations)[i].begin);



		// Read the number of photons
		ii=0;
		while(ii<=0)
		{
			fgets(mystring , STR_LEN , pFile);
			number_of_photons=0;
			ii=sscanf(mystring,"%lu",&number_of_photons);
			if(feof(pFile) || ii>1){perror("Error reading number of photons");return 0;} //if we reach EOF or read more number than defined something is wrong with the file!
			//printf("ii=%d temp=%f %f %f %f %f\n",ii,temp[0],temp[1],temp[2],temp[3],temp[4]);
		}
		//printf("Number of photons: %lu\n",number_of_photons);
		(*simulations)[i].number_of_photons=number_of_photons;

		// Read dr and dz (3x float)
		if(!readfloats(2, ftemp, pFile)){perror ("Error reading dr and dz");return 0;}
		//printf("dz=%f, dx=%f, dy=%f\n",ftemp[0],ftemp[1],ftemp[2]);
		//(*simulations)[i].det.dz=ftemp[0];
		(*simulations)[i].det.dx=ftemp[0];
		(*simulations)[i].det.dy=ftemp[1];
		
		// Read No. of dz, dr and da  (3x int)
		if(!readints(2, itemp, pFile)){perror ("Error reading No. of dz, dr and da");return 0;}

		//printf("No. of dz=%d, dx=%d, dy=%d\n",itemp[0],itemp[1],itemp[2]);
		//(*simulations)[i].det.nz=itemp[0];
		(*simulations)[i].det.nx=itemp[0];
		(*simulations)[i].det.ny=itemp[1];

		// Leer separacion fuente-detector	
		//if(!readfloats(1, ftemp, pFile)){perror ("Error leyendo separacion fuente-detector");return 0;}
		//printf("Useparacion fuente-detector=%f\n",ftemp[0]);
		//(*simulations)[i].det.sep=ftemp[0];
		
		// Leer posicion de la fibra (3x float)
		if(!readfloats(3, ftemp, pFile)){perror ("Error leyendo fix and fiy");return 0;}
		(*simulations)[i].det.face=(int)ftemp[0];
		(*simulations)[i].det.fix=ftemp[1];
		(*simulations)[i].det.fiy=ftemp[2];
		
		printf("Detector: cara=%u, x=%f, y=%f\n",(*simulations)[i].det.face,(*simulations)[i].det.fix,(*simulations)[i].det.fiy);

		
		
		// Leer posicion de la fuente (2x float)
		if(!readfloats(2, ftemp, pFile)){perror ("Error leyendo fx and fy");return 0;}
		(*simulations)[i].fx=ftemp[0];
		(*simulations)[i].fy=ftemp[1];
		printf("Fuente: x=%f, y=%f\n",(*simulations)[i].fx,(*simulations)[i].fy);


		// Read No. of layers (1xint)
		if(!readints(1, itemp, pFile)){perror ("Error reading No. of layers");return 0;}
		printf("No. of layers=%d\n",itemp[0]);
		n_layers = itemp[0];
		(*simulations)[i].n_layers = itemp[0];
		printf("No. of layers of %i = %d\n",i,(*simulations)[i].n_layers);

		// Allocate memory for the layers (including one for the upper and one for the lower)
		(*simulations)[i].layers = (LayerStruct*) malloc(sizeof(LayerStruct)*(n_layers+2));
		if((*simulations)[i].layers == NULL){perror("Failed to malloc layers.\n");return 0;}//{printf("Failed to malloc simulations.\n");return 0;}


		// Read upper refractive index (1xfloat)
		if(!readfloats(1, ftemp, pFile)){perror ("Error reading upper refractive index");return 0;}
		printf("Upper refractive index=%f\n",ftemp[0]);
		(*simulations)[i].layers[0].n=ftemp[0];

		dtot=0;
		for(ii=1;ii<=n_layers;ii++)
		{
			// Read Layer data (5x float)
			if(!readfloats(5, ftemp, pFile)){perror ("Error reading layer data");return 0;}
			printf("n=%f, mua=%f, mus=%f, g=%f, d=%f\n",ftemp[0],ftemp[1],ftemp[2],ftemp[3],ftemp[4]);
			(*simulations)[i].layers[ii].n=ftemp[0];
			(*simulations)[i].layers[ii].mua=ftemp[1];
			(*simulations)[i].layers[ii].g=ftemp[3];
			(*simulations)[i].layers[ii].z_min=dtot;
			dtot+=ftemp[4];
			(*simulations)[i].layers[ii].z_max=dtot;
			if(ftemp[2]==0.0f)(*simulations)[i].layers[ii].mutr=FLT_MAX; //Glass layer
			else(*simulations)[i].layers[ii].mutr=1.0f/(ftemp[1]+ftemp[2]);
		}//end ii<n_layers
		
		//Calcular espesor
		printf("Espesor=%f\n",dtot);
		(*simulations)[i].esp=dtot;
		
		// Read lower refractive index (1xfloat)
		if(!readfloats(1, ftemp, pFile)){perror ("Error reading lower refractive index");return 0;}
		printf("Lower refractive index=%f\n",ftemp[0]);
		(*simulations)[i].layers[n_layers+1].n=ftemp[0];            
		
		// Read number of inclusions (1xint)
		if(!readints(1, itemp, pFile)){perror ("Error leyendo el numero de inclusiones");return 0;}
		printf("Number of inclusions=%d\n",itemp[0]);
		n_inclusions = itemp[0];
		(*simulations)[i].n_inclusions = itemp[0];
		
		// Allocate memory for the inclusions 
		(*simulations)[i].inclusion = (IncStruct*) malloc(sizeof(IncStruct)*(n_inclusions));
		if((*simulations)[i].inclusion == NULL){perror("Failed to malloc inclusions.\n");return 0;}                                                                             

		// Read inclusion data (9xfloat)
		for(ii=0;ii<n_inclusions;ii++)	{
			if(!readfloats(9, ftemp, pFile)){perror ("Error leyendo datos de inclusion");return 0;}
			printf("type= %f, x=%f, y=%f, z=%f, r=%f, n=%f, mua=%f, mus=%f, g=%f\n",ftemp[0],ftemp[1],ftemp[2],ftemp[3],ftemp[4],ftemp[5],ftemp[6],ftemp[7],ftemp[8]);
			(*simulations)[i].inclusion[ii].type=(int)ftemp[0];
			(*simulations)[i].inclusion[ii].x=ftemp[1];
			(*simulations)[i].inclusion[ii].y=ftemp[2];
			(*simulations)[i].inclusion[ii].z=ftemp[3];
			(*simulations)[i].inclusion[ii].r=ftemp[4];
			(*simulations)[i].inclusion[ii].n=ftemp[5];
			(*simulations)[i].inclusion[ii].mua=ftemp[6];
			(*simulations)[i].inclusion[ii].g=ftemp[8];
			if(ftemp[7]==0.0f)(*simulations)[i].inclusion[ii].mutr=FLT_MAX; //Inclusion with mus=0
			//else(*simulations)[i].inclusion.mutr=1.0f/(ftemp[4]+ftemp[5]);
			else //Calculates the corrected mus and mutr
			{
				for (iii=1; iii<=(*simulations)[i].n_layers; iii++){
					float z_min_temp = (*simulations)[i].layers[iii].z_min;
					printf ("iii %i , z min %f\n",iii,z_min_temp);
					float z_max_temp = (*simulations)[i].layers[iii].z_max;
					printf ("iii %i , z max %f\n",iii,z_max_temp);
					float z_temp = ftemp[3];
					printf ("iii %i , z %f\n",iii,z_temp);
					if (z_temp>z_min_temp && z_temp<=z_max_temp){
						(*simulations)[i].inclusion[ii].layer = iii;
						float corr=((*simulations)[i].layers[iii].n/ftemp[5])*((*simulations)[i].layers[iii].n/ftemp[5]);
						printf ("layer= %u, nmed: %f, ninc: %f, corr: %f\n",(*simulations)[i].inclusion[ii].layer,(*simulations)[i].layers[iii].n,ftemp[4],corr);
						(*simulations)[i].inclusion[ii].mutr=1.0f/(ftemp[6]+corr*ftemp[7]);
					}
				}
			}	
		
			printf("inclusion %i: type:%u, x=%f, y=%f, z=%f, r=%f, n=%f, mua=%f, mutr=%f, g=%f\n",ii,(*simulations)[i].inclusion[ii].type,(*simulations)[i].inclusion[ii].x,(*simulations)[i].inclusion[ii].y,(*simulations)[i].inclusion[ii].z,(*simulations)[i].inclusion[ii].r,(*simulations)[i].inclusion[ii].n,(*simulations)[i].inclusion[ii].mua,(*simulations)[i].inclusion[ii].mutr,(*simulations)[i].inclusion[ii].g);
		
		}	
		

		(*simulations)[i].end=ftell(pFile);
		//printf("end=%d\n",(*simulations)[i].end);
		
		
		
		//calculate start_weight
		double n1=(*simulations)[i].layers[0].n;
		double n2=(*simulations)[i].layers[1].n;
		double r = (n1-n2)/(n1+n2);
		r = r*r;
		start_weight = (unsigned int)((double)0xffffffff*(1-r));
		//printf("Start weight=%u\n",start_weight);
		(*simulations)[i].start_weight=start_weight;
		

	}//end for i<n_simulations
	return n_simulations;
}
