#include "hip/hip_runtime.h"
/*	This file is part of CUDAMCML_INC.

    CUDAMCML_INC is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCML_INC is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCML_INC.  If not, see <http://www.gnu.org/licenses/>.*/

// forward declaration of the device code
__global__ void MCd(MemStruct);
__device__ float rand_MWC_oc(unsigned long long*,unsigned int*);
__device__ float rand_MWC_co(unsigned long long*,unsigned int*);
__device__ void LaunchPhoton(PhotonStruct*,unsigned long long*, unsigned int*, MemStruct);
__global__ void LaunchPhoton_Global(MemStruct);
__device__ void Spin(PhotonStruct*, float,unsigned long long*,unsigned int*);
__device__ unsigned int Reflect(PhotonStruct*, int, unsigned long long*, unsigned int*);
__device__ unsigned int PhotonSurvive(PhotonStruct*, unsigned long long*, unsigned int*);
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add);

__global__ void MCd(MemStruct DeviceMem)
{
    //Block index
    int bx=blockIdx.x;

    //Thread index
    int tx=threadIdx.x;

	int num_hilo=bx*NUM_THREADS_PER_BLOCK+tx;	
	//printf("%i\n",num_hilo);
	

    //First element processed by the block
    int begin=NUM_THREADS_PER_BLOCK*bx;
	
	float esp=layers_dc[(*n_layers_dc)].z_max;	
	unsigned int max_z = __float2uint_rz(esp*TAM_GRILLA);
	
	unsigned long long int x=DeviceMem.x[begin+tx];//coherent
	unsigned int a=DeviceMem.a[begin+tx];//coherent

	float s;	//step length
	//int index_x,index_y;
	int index;
	int b_index;
	double temp;
	int temp_index;
	unsigned int w;
	
	PhotonStruct p = DeviceMem.p[begin+tx];


	int new_layer;

	//float __fdividef(det_dc[0].dx*det_dc[0].nx,2)=__fdividef(det_dc[0].dx*det_dc[0].nx,2);
	//float (-(__fdividef(det_dc[0].dx*det_dc[0].nx,2)))=-(__fdividef(det_dc[0].dx*det_dc[0].nx,2));
	//float __fdividef(det_dc[0].dy*det_dc[0].ny,2)=__fdividef(det_dc[0].dy*det_dc[0].ny,2);
	//float (-(__fdividef(det_dc[0].dy*det_dc[0].ny,2)))=-(__fdividef(det_dc[0].dy*det_dc[0].ny,2));
	//float max_det_z=__fdividef(det_dc[0].dz*det_dc[0].nz,2);
	//float min_det_z=-(__fdividef(det_dc[0].dz*det_dc[0].nz,2));

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if(!DeviceMem.thread_active[begin+tx]) ii = NUMSTEPS_GPU;

	for(;ii<NUMSTEPS_GPU;ii++) //this is the main while loop
	{
		
		if(p.step<MAX_STEP){
			if(fabsf(p.x)<esp) DeviceMem.xx[p.step*NUM_THREADS+num_hilo]= __float2uint_rz((p.x+esp)*TAM_GRILLA);
				else DeviceMem.xx[p.step*NUM_THREADS+num_hilo] = __float2uint_rz((2*esp)*TAM_GRILLA);
		
			if(p.z<esp) DeviceMem.zz[p.step*NUM_THREADS+num_hilo]= __float2uint_rz((p.z)*TAM_GRILLA);
				else DeviceMem.zz[p.step*NUM_THREADS+num_hilo]= __float2uint_rz(esp*TAM_GRILLA);
		}
	
		if(p.inc<0)
		{// Outside of inclusions		
			if(layers_dc[p.layer].mutr!=FLT_MAX)
				s = -__logf(rand_MWC_oc(&x,&a))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
			else
				s = GLASS_STEP;//temporary, say the step in glass is 100 cm.
		}
		else
		{// Inside inclusion number inc
			if(inclusion_dc[p.inc].mutr!=FLT_MAX)
				s = -__logf(rand_MWC_oc(&x,&a))*(inclusion_dc[p.inc].mutr);//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
			else
				s = GLASS_STEP;//temporary, say the step in glass is 100 cm.
		}
		p.inc = -1; // Set inc to default value, outside inclusions
		
		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if(p.z+s*p.dz<layers_dc[p.layer].z_min){new_layer--; s = __fdividef(layers_dc[p.layer].z_min-p.z,p.dz);} //Check for upwards reflection/transmission & calculate new s
		if(p.z+s*p.dz>layers_dc[p.layer].z_max){new_layer++; s = __fdividef(layers_dc[p.layer].z_max-p.z,p.dz);} //Check for downward reflection/transmission
		
		

		p.t += fabsf(s)/(LIGHTSPEED/layers_dc[p.layer].n);
		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;
		if (p.step<(MAX_STEP-1)) p.step ++;
		
		
		
		if(new_layer!=p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;  
 
			if(Reflect(&p,new_layer,&x,&a)==0u)//Check for reflection
			{ // Photon is transmitted
				if(new_layer == 0)
				{ //Diffuse reflectance
										
					//index = __float2int_rz(acosf(-p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr+min(__float2int_rz(__fdividef(sqrtf(p.x*p.x+p.y*p.y),det_dc[0].dr)),(int)det_dc[0].nr-1);
					
					if(det_dc[0].face==1){//Detector in entry face
						if(((p.x-det_dc[0].fix)*(p.x-det_dc[0].fix)+(p.y-det_dc[0].fiy)*(p.y-det_dc[0].fiy))<=(RAD_FIB_BAN*RAD_FIB_BAN)){
						
						// Temporal Histogram
						
							temp_index =  __float2int_rz(p.t/TEMP_CAN);
   							if (temp_index >= NUM_CAN_TEMP) temp_index=NUM_CAN_TEMP-1;
							if (temp_index < 0) temp_index=0;
							AtomicAddULL(&DeviceMem.histo_temp[temp_index],p.weight);
						
						// Banana
						
							for (int i=0; i<p.step; i++){
								if((fabsf(DeviceMem.xx[i])<esp)||(DeviceMem.zz[i]<esp)){							
									b_index = DeviceMem.xx[i*NUM_THREADS+num_hilo]*max_z+DeviceMem.zz[i*NUM_THREADS+num_hilo];
									AtomicAddULL(&DeviceMem.banana[b_index], p.weight);
								//printf ("%i, %i, %lld\n",i , b_index, DeviceMem.banana[b_index]);
								}				 
							}
						}
					}
					
					if(p.x<__fdividef(det_dc[0].dx*det_dc[0].nx,2) && p.x>(-(__fdividef(det_dc[0].dx*det_dc[0].nx,2))) && p.y<__fdividef(det_dc[0].dy*det_dc[0].ny,2) && p.y>(-(__fdividef(det_dc[0].dy*det_dc[0].ny,2))))
					{//Inside the detector
						// Calculates the position in the exit matrix			
						//index=(__float2int_rz(fabsf(__fdividef(p.x+__fdividef(det_dc[0].dx*det_dc[0].nx,2),det_dc[0].dx))))*det_dc[0].ny+__float2int_rz(fabsf(__fdividef(p.y+__fdividef(det_dc[0].dy*det_dc[0].ny,2),det_dc[0].dy)));
						index=(__float2int_rn(fabsf(__fdividef(p.y+__fdividef(det_dc[0].dy*det_dc[0].ny,2),det_dc[0].dy))))*det_dc[0].nx+__float2int_rn(fabsf(__fdividef(p.x+__fdividef(det_dc[0].dx*det_dc[0].nx,2),det_dc[0].dx)));
						AtomicAddULL(&DeviceMem.Rd_xy[index], p.weight);
					}
				p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
				if(new_layer > *n_layers_dc)
				{	//Transmitted
					//index = __float2int_rz(acosf(p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr+min(__float2int_rz(__fdividef(sqrtf(p.x*p.x+p.y*p.y),det_dc[0].dr)),(int)det_dc[0].nr-1);
					if(det_dc[0].face==0){//Detector in exit face
						if(((p.x-det_dc[0].fix)*(p.x-det_dc[0].fix)+(p.y-det_dc[0].fiy)*(p.y-det_dc[0].fiy))<=(RAD_FIB_BAN*RAD_FIB_BAN)){
						
						// Temporal Histogram
						
							temp_index =  __float2int_rz(p.t/TEMP_CAN);
   							if (temp_index >= NUM_CAN_TEMP) temp_index=NUM_CAN_TEMP-1;
							if (temp_index < 0) temp_index=0;
							AtomicAddULL(&DeviceMem.histo_temp[temp_index],p.weight);
						
						// Banana
						
							for (int i=0; i<p.step; i++){
								if((fabsf(DeviceMem.xx[i])<esp)||(DeviceMem.zz[i]<esp)){							
									b_index = DeviceMem.xx[i*NUM_THREADS+num_hilo]*max_z+DeviceMem.zz[i*NUM_THREADS+num_hilo];
									AtomicAddULL(&DeviceMem.banana[b_index], p.weight);
								//printf ("%i, %i, %lld\n",i , b_index, DeviceMem.banana[b_index]);
								}				 
							}
						}
					}
					if(fabsf(p.x)<__fdividef(det_dc[0].dx*det_dc[0].nx,2) && fabsf(p.y)<__fdividef(det_dc[0].dy*det_dc[0].ny,2))
					{//Estoy dentro del detector
						// Calculates the position in the exit matrix				
						//index=(__float2int_rz(fabsf(__fdividef(p.x+__fdividef(det_dc[0].dx*det_dc[0].nx,2),det_dc[0].dx))+0.5))*det_dc[0].ny+__float2int_rz(fabsf(__fdividef(p.y+__fdividef(det_dc[0].dy*det_dc[0].ny,2),det_dc[0].dy))+0.5);
						index=(__float2int_rn(fabsf(__fdividef(p.y+__fdividef(det_dc[0].dy*det_dc[0].ny,2),det_dc[0].dy))))*det_dc[0].nx+__float2int_rn(fabsf(__fdividef(p.x+__fdividef(det_dc[0].dx*det_dc[0].nx,2),det_dc[0].dx)));
						AtomicAddULL(&DeviceMem.Tt_xy[index], p.weight);
					}
					//AtomicAddULL(&DeviceMem.Tt_xy[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		w=0;
		
		if(s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			//if((p.layer == inclusion_dc[0].layer)&&((((p.x-inclusion_dc[0].x)*(p.x-inclusion_dc[0].x))+((p.y-inclusion_dc[0].y)*(p.y-inclusion_dc[0].y))+((p.z-inclusion_dc[0].z)*(p.z-inclusion_dc[0].z)))<=(inclusion_dc[0].r*inclusion_dc[0].r)))
			
			int inc=0;
			
			while (inc<*n_inclusions_dc && p.inc<0)			
			{
				if( ( (inclusion_dc[inc].type==1) && (p.layer == inclusion_dc[inc].layer) && ((((p.x-inclusion_dc[inc].x)*(p.x-inclusion_dc[inc].x))+((p.z-inclusion_dc[inc].z)*(p.z-inclusion_dc[inc].z)))<=(inclusion_dc[inc].r*inclusion_dc[inc].r)) ) || ( (inclusion_dc[inc].type==0) && (p.layer == inclusion_dc[inc].layer) && ((((p.x-inclusion_dc[inc].x)*(p.x-inclusion_dc[inc].x))+((p.y-inclusion_dc[inc].y)*(p.y-inclusion_dc[inc].y))+((p.z-inclusion_dc[inc].z)*(p.z-inclusion_dc[inc].z)))<=(inclusion_dc[inc].r*inclusion_dc[inc].r)) ) )
				{// Estamos dentro de la inclusion
				//printf("entre \n");
				w = __float2uint_rn(inclusion_dc[inc].mua*inclusion_dc[inc].mutr*__uint2float_rn(p.weight));
				p.weight -= w;//__int_as_float(data.w);
				p.inc = inc;	
				Spin(&p,inclusion_dc[inc].g,&x,&a);
				}
				else{	
					inc++;
				}
			
			
			}	
			
			if (p.inc<0) {
				w = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
				p.weight -= w;//__int_as_float(data.w);
				Spin(&p,layers_dc[p.layer].g,&x,&a);
			}
		
		}
		
		w = w&(*ignoreAdetection_dc); //this will set w to 0 if user has specified to ignore detection of absorbed weight

		//if(w!=0u) AtomicAddULL(&DeviceMem.A_xyz[index], w);


		if(!PhotonSurvive(&p,&x,&a)) // Check if photons survives or not
		{
			if(atomicAdd(DeviceMem.num_terminated_photons,1u) < (*num_photons_dc-NUM_THREADS))
			{	// Ok to launch another photon
				LaunchPhoton(&p,&x,&a, DeviceMem);//Launch a new photon
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[begin+tx] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}
			
		}
	}//end main for loop!
	__syncthreads();//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[begin+tx] = p;	//This one is incoherent!!!
	DeviceMem.x[begin+tx] = x; //this one also seems to be coherent
	
	

}//end MCd




__device__ void LaunchPhoton(PhotonStruct* p, unsigned long long* x, unsigned int* a, MemStruct DeviceMem)
{
	// We are currently not using the RNG but might do later
	//float input_fibre_radius = 0.03;//[cm]
	//p->x=input_fibre_radius*sqrtf(rand_MWC_co(x,a));

	p->x  = (float)*fx_dc;
	p->y  = (float)*fy_dc;
	p->z  = 0.0f;
	p->t  = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;
	p->step= 0;
	/*for (int i =0; i<MAX_STEP; i++){
		DeviceMem.xx[i]=0;
		DeviceMem.zz[i]=0;
	}*/
	p->inc=-1;
	
	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}

__global__ void LaunchPhoton_Global(MemStruct DeviceMem)//PhotonStruct* pd, unsigned long long* x, unsigned int* a)
{
	int bx=blockIdx.x;
    	int tx=threadIdx.x;	

    	//First element processed by the block
    	int begin=NUM_THREADS_PER_BLOCK*bx;

	PhotonStruct p;
	unsigned long long int x=DeviceMem.x[begin+tx];//coherent

	unsigned int a=DeviceMem.a[begin+tx];//coherent

	LaunchPhoton(&p,&x,&a, DeviceMem);

	//__syncthreads();//necessary?
	DeviceMem.p[begin+tx]=p;//incoherent!?
}


__device__ void Spin(PhotonStruct* p, float g, unsigned long long* x, unsigned int* a)
{
	float cost, sint;	// cosine and sine of the 
						// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
						// azimuthal angle psi. 
	float temp;

	float tempdir=p->dx;

	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f-(g)*(g)),(1.0f-(g)+2.0f*(g)*rand_MWC_co(x,a)));//Should be close close????!!!!!
	cost = __fdividef((1.0f+(g)*(g) - temp*temp),(2.0f*(g)));
	if(g==0.0f)
		cost = 2.0f*rand_MWC_co(x,a) -1.0f;//Should be close close??!!!!!

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x,a),&cosp,&sinp);// spin psi [0-2*PI)
	
	temp = sqrtf(1.0f - p->dz*p->dz);

	if(temp==0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost,p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp),temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp),temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp=rsqrtf(p->dx*p->dx+p->dy*p->dy+p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin

			

__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, unsigned long long* x, unsigned int* a)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)


	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if(n1==n2)//refraction index matching automatic transmission and no direction change
	{	
		p->layer = new_layer;
		return 0u;
	}

	if(n2*n2<n1*n1*(1-cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u; 
	}

	if(cos_angle_i==1.0f)//normal incident
	{		
		r = __fdividef((n1-n2),(n1+n2));
		if(rand_MWC_co(x,a)<=r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}
	
	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	float e = __fdividef(n1*n1,n2*n2)*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	r=2*sqrtf((1.0f-cos_angle_i*cos_angle_i)*(1.0f-e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	e=e+(cos_angle_i*cos_angle_i)*(1.0f-2.0f*e);//Update the value of e
	r = e*__fdividef((1.0f-e-r),((1.0f-e+r)*(e+r)));//Calculate r	

	if(rand_MWC_co(x,a)<=r)
	{ 
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{	
		// Transmission, update layer and direction
		r = __fdividef(n1,n2);
		e = r*r*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1-e) ,p->dz);
		p->layer = new_layer;
		return 0u;
	}

}

__device__ unsigned int PhotonSurvive(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if(p->weight>WEIGHTI) return 1u; // No roulette needed
	if(p->weight==0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if(rand_MWC_co(x,a)<CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight,CHANCE));
		return 1u;
	}

	//else
	return 0u;
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add)
{
	if(atomicAdd((unsigned int*)address,add)+add<add)
		atomicAdd(((unsigned int*)address)+1,1u);
}
